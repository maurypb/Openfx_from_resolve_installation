// TestBlurV2.cu - Generated skeleton
// Fill in the image processing logic in the kernel function

#include <hip/hip_runtime.h>
#include <cmath>

// Image processing kernel - implement your algorithm here
__global__ void TestBlurV2Kernel(
    int width,
    int height,
    hipTextureObject_t inputTex,
    hipTextureObject_t maskTex,
    float* output,
    bool maskPresent,
    // Auto-generated from XML parameters:
    float brightness,  // from <parameter name="brightness" type="double">
    float radius,  // from <parameter name="radius" type="double">
    int quality,  // from <parameter name="quality" type="int">
    float maskStrength  // from <parameter name="maskStrength" type="double">
)
{
    // Standard CUDA coordinate calculation
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
   // Normalize coordinates to [0,1] range for texture fetch
   float u = (x + 0.5f) / width;
   float v = (y + 0.5f) / height;
 
    if ((x < width) && (y < height))
    {
        // Output index
        const int index = ((y * width) + x) * 4;
        
        // Read mask value if available
        float maskValue = 1.0f;  // Default to full blur 
        
        // Sample directly from source image
        float4 srcColor = tex2D<float4>(inputTex, u, v);
        
        if (maskPresent) {
            if (maskStrength >= 0.0f) {
                // Sample mask and apply (includes 0.0f case)
                float4 maskColor = tex2D<float4>(maskTex, u, v);
                maskValue = maskStrength * maskColor.w;
            } else {
                // Negative mask strength = no blur at all
                maskValue = 0.0f;
            }
        }
 
        // Calculate effective blur radius based on mask
        float effectiveRadius = radius * maskValue;
        
        // Early exit if no blur needed (either radius is 0 or mask is 0)
        if (effectiveRadius <= 0.0f) {
            // Just copy the source pixel - no blur applied
            output[index + 0] = srcColor.x;
            output[index + 1] = srcColor.y;
            output[index + 2] = srcColor.z;
            output[index + 3] = srcColor.w;
            return;
        }
        
        // Gaussian blur implementation
        float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        float weightSum = 0.0f;
        
        // Perform sampling in a circle
        for (int i = 0; i < quality; ++i) {
            // Calculate sample angle
            float angle = (2.0f * 3.14159f * i) / quality;
            
            // Calculate sample positions at different distances
            for (float distance = 1.0f; distance <= effectiveRadius; distance += 1.0f) {
                float sampleX = x + cos(angle) * distance;
                float sampleY = y + sin(angle) * distance;
                
                // Normalize coordinates to [0,1] range
                float sample_u = (sampleX + 0.5f) / width;
                float sample_v = (sampleY + 0.5f) / height;
                
                // Sample using texture
                float4 color = tex2D<float4>(inputTex, sample_u, sample_v);
                
                // Calculate weight (simplified for now)
                float weight = 1.0f;
                
                // Accumulate weighted color
                sum.x += color.x * weight;
                sum.y += color.y * weight;
                sum.z += color.z * weight;
                sum.w += color.w * weight;
                weightSum += weight;
            }
        }
        
        // Normalize by total weight
        if (weightSum > 0.0f) {
            sum.x /= weightSum;
            sum.y /= weightSum;
            sum.z /= weightSum;
            sum.w /= weightSum;
        }
        
        // Write to output
        output[index + 0] = sum.x;
        output[index + 1] = sum.y;
        output[index + 2] = sum.z;
        output[index + 3] = sum.w;
    }
}

// Bridge function - connects framework to your kernel
extern "C" void call_testblurv2_kernel(
    void* stream, int width, int height,
    float brightness,
    float radius,
    int quality,
    float maskStrength,
    hipTextureObject_t inputTex, hipTextureObject_t maskTex,
    float* output, bool maskPresent
) {
    hipStream_t cudaStream = static_cast<hipStream_t>(stream);

    // Launch configuration
    dim3 threads(16, 16, 1);
    dim3 blocks(((width + threads.x - 1) / threads.x), ((height + threads.y - 1) / threads.y), 1);

    // Launch the kernel
    TestBlurV2Kernel<<<blocks, threads, 0, cudaStream>>>(
        width, height, inputTex, maskTex, output, maskPresent,
        brightness,
        radius,
        quality,
        maskStrength
    );
}
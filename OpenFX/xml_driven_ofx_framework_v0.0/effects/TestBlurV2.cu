// TestBlurV2.cu - Generated skeleton
// Fill in the image processing logic in the kernel function

#include <hip/hip_runtime.h>
#include <cmath>

// Image processing kernel - implement your algorithm here
__global__ void TestBlurV2Kernel(
    int width,
    int height,
    hipTextureObject_t SourceTex,  // from <source name="Source" optional="False" border_mode="clamp">
    hipTextureObject_t maskTex,  // from <source name="mask" optional="True" border_mode="black">
    bool maskPresent,  // whether mask is connected
    hipTextureObject_t selectiveTex,  // from <source name="selective" optional="True" border_mode="black">
    bool selectivePresent,  // whether selective is connected
    float* output,
    float brightness,  // from <parameter name="brightness" type="double" default="1.0">
    float radius,  // from <parameter name="radius" type="double" default="30.0">
    int quality,  // from <parameter name="quality" type="int" default="8">
    float maskStrength  // from <parameter name="maskStrength" type="double" default="1.0">
)
{
    // Standard CUDA coordinate calculation
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height)) {
        // Normalize coordinates to [0,1] range for texture sampling
        float u = (x + 0.5f) / width;
        float v = (y + 0.5f) / height;
        // Calculate output array index
        const int index = ((y * width) + x) * 4;

    
        // Read mask value if available
        float maskValue = 1.0f;  // Default to full blur 
        
        // Sample directly from source image
        float4 srcColor = tex2D<float4>(SourceTex, u, v);
        
        if (maskPresent) {
            if (maskStrength >= 0.0f) {
                // Sample mask and apply (includes 0.0f case)
                float4 maskColor = tex2D<float4>(maskTex, u, v);
                maskValue = maskStrength * maskColor.w;
            } else {
                // Negative mask strength = no blur at all
                maskValue = 0.0f;
            }
        }

        // Calculate effective blur radius based on mask
        float effectiveRadius = radius * maskValue;
        
        // Early exit if no blur needed (either radius is 0 or mask is 0)
        if (effectiveRadius <= 0.0f) {
            // Just copy the source pixel - no blur applied
            output[index + 0] = srcColor.x*brightness;
            output[index + 1] = srcColor.y*brightness;
            output[index + 2] = srcColor.z*brightness;
            output[index + 3] = srcColor.w;
            return;
        }
        
        // Gaussian blur implementation
        float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        float weightSum = 0.0f;
        
        // Perform sampling in a circle
        for (int i = 0; i < quality; ++i) {
            // Calculate sample angle
            float angle = (2.0f * 3.14159f * i) / quality;
            
            // Calculate sample positions at different distances
            for (float distance = 1.0f; distance <= effectiveRadius; distance += 1.0f) {
                float sampleX = x + cos(angle) * distance;
                float sampleY = y + sin(angle) * distance;
                
                // Normalize coordinates to [0,1] range
                float sample_u = (sampleX + 0.5f) / width;
                float sample_v = (sampleY + 0.5f) / height;
                
                // Sample using texture
                float4 color = tex2D<float4>(SourceTex, sample_u, sample_v);
                
                // Calculate weight (simplified for now)
                float weight = 1.0f;
                
                // Accumulate weighted color
                sum.x += color.x * weight;
                sum.y += color.y * weight;
                sum.z += color.z * weight;
                sum.w += color.w * weight;
                weightSum += weight;
            }
        }
        
        // Normalize by total weight
        if (weightSum > 0.0f) {
            sum.x /= weightSum;
            sum.y /= weightSum;
            sum.z /= weightSum;
            sum.w /= weightSum;
        }
        sum.x = sum.x * brightness;
        sum.y=sum.y * brightness;
        sum.z=sum.z*brightness;
 
        // Write to output
        output[index + 0] = sum.x;
        output[index + 1] = sum.y;
        output[index + 2] = sum.z;
        output[index + 3] = sum.w;


    }
}

// Bridge function - connects framework to your kernel
extern "C" void call_testblurv2_kernel(
    void* stream, int width, int height,
    hipTextureObject_t SourceTex,
    hipTextureObject_t maskTex,
    bool maskPresent,
    hipTextureObject_t selectiveTex,
    bool selectivePresent,
    float* output,
    float brightness,
    float radius,
    int quality,
    float maskStrength
) {
    hipStream_t cudaStream = static_cast<hipStream_t>(stream);

    // Launch configuration
    dim3 threads(16, 16, 1);
    dim3 blocks(((width + threads.x - 1) / threads.x), ((height + threads.y - 1) / threads.y), 1);

    // Launch the kernel
    TestBlurV2Kernel<<<blocks, threads, 0, cudaStream>>>(
        width,
        height,
        SourceTex,
        maskTex,
        maskPresent,
        selectiveTex,
        selectivePresent,
        output,
        brightness,
        radius,
        quality,
        maskStrength
    );
}